#include "hip/hip_runtime.h"
// src/cpp/data_generator_cuda.cu

#include <hip/hip_runtime.h>
#include <>
#include <cmath>
#include <map>
#include <vector>
#include <iostream>
#include "data_generator.h"

// 自定义的 atomicAdd 函数，用于 double 类型
__device__ double atomicAddDouble(double *address, double value)
{
    unsigned long long int *address_as_ull =
        (unsigned long long int *)address;
    unsigned long long int old = *address_as_ull, assumed;
    do
    {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(value + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

// CUDA 核函数用于计算平均信号功率
__global__ void avg_signal_pw_kernel(double *d_total_pw, const double *d_distances, int nPU, int nSU, double beta, double alpha, double DistAmp, int nBandsPerPU)
{
    int suIdx = blockIdx.x * blockDim.x + threadIdx.x;
    if (suIdx < nSU)
    {
        for (int PU = 0; PU < nPU; ++PU)
        {
            double distance = DistAmp * d_distances[PU * nSU + suIdx];
            double gain = 1.0 / (beta * pow(distance, alpha));
            double power = nBandsPerPU * gain;

            // 使用自定义的 atomicAddDouble 函数避免数据竞争
            atomicAddDouble(d_total_pw, power);
        }
    }
}

// 主机函数，用于调用 CUDA 核函数计算平均信号功率
double avg_signal_pw_cuda(
    const std::map<int, std::vector<double>> &dist_dict,
    double beta,
    double alpha,
    int nPU,
    int nSU,
    int nch,
    double DistAmp,
    int nBandsPerPU)
{
    // 分配和初始化设备内存
    double *d_distances;
    hipMalloc((void **)&d_distances, nPU * nSU * sizeof(double));

    double *distances = new double[nPU * nSU];
    for (int PU = 0; PU < nPU; ++PU)
    {
        for (int SU = 0; SU < nSU; ++SU)
        {
            distances[PU * nSU + SU] = dist_dict.at(PU)[SU];
        }
    }
    hipMemcpy(d_distances, distances, nPU * nSU * sizeof(double), hipMemcpyHostToDevice);

    double total_pw = 0.0;
    double *d_total_pw;
    hipMalloc((void **)&d_total_pw, sizeof(double));
    hipMemcpy(d_total_pw, &total_pw, sizeof(double), hipMemcpyHostToDevice);

    // 启动 CUDA 核函数
    int threadsPerBlock = 256;
    int blocksPerGrid = (nSU + threadsPerBlock - 1) / threadsPerBlock;
    avg_signal_pw_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_total_pw, d_distances, nPU, nSU, beta, alpha, DistAmp, nBandsPerPU);

    // 同步并获取结果
    hipMemcpy(&total_pw, d_total_pw, sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    double avg_pw = total_pw / (nSU * nch);

    // 清理设备内存
    hipFree(d_distances);
    hipFree(d_total_pw);
    delete[] distances;

    return avg_pw;
}

// CUDA 核函数用于生成数据
__global__ void generate_data_kernel(double *d_data, double *d_noises, const double *d_distances, int nPU, int nSU, int nch, int nw, double beta, double alpha, double DistAmp, double noi_pw)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int totalElements = nSU * nch * nw;
    if (idx < totalElements)
    {
        int suIdx = idx / (nch * nw);
        int chIdx = (idx / nw) % nch;
        int sampleIdx = idx % nw;

        // 加入噪声
        d_data[idx] = d_noises[idx];

        for (int PU = 0; PU < nPU; ++PU)
        {
            double distance = DistAmp * d_distances[PU * nSU + suIdx];
            double ch_gain = 1.0 / (beta * pow(distance, alpha));

            // 对信道增益进行阴影衰落处理（此处可加入随机衰落，但为简单起见省略）
            // ch_gain *= shadow_fading_factor;

            // 将信号加权并添加到数据中
            d_data[idx] += ch_gain; // 简化版，仅为演示
        }
    }
}

// 主机函数，用于调用 CUDA 核函数生成数据
std::vector<std::vector<std::vector<double>>> generate_data_cuda(
    const std::map<int, std::vector<double>> &dist_dict,
    double beta,
    double alpha,
    int nPU,
    int nSU,
    int nch,
    int nw,
    double DistAmp,
    double noi_pw)
{

    // 分配和初始化设备内存
    double *d_distances;
    hipMalloc((void **)&d_distances, nPU * nSU * sizeof(double));

    double *distances = new double[nPU * nSU];
    for (int PU = 0; PU < nPU; ++PU)
    {
        for (int SU = 0; SU < nSU; ++SU)
        {
            distances[PU * nSU + SU] = dist_dict.at(PU)[SU];
        }
    }
    hipMemcpy(d_distances, distances, nPU * nSU * sizeof(double), hipMemcpyHostToDevice);

    double *d_data;
    hipMalloc((void **)&d_data, nSU * nch * nw * sizeof(double));

    double *d_noises;
    hipMalloc((void **)&d_noises, nSU * nch * nw * sizeof(double));

    // 随机初始化噪声数据
    double *noises = new double[nSU * nch * nw];
    for (int i = 0; i < nSU * nch * nw; ++i)
    {
        noises[i] = noi_pw * ((double)rand() / RAND_MAX);
    }
    hipMemcpy(d_noises, noises, nSU * nch * nw * sizeof(double), hipMemcpyHostToDevice);

    // 启动 CUDA 核函数生成数据
    int threadsPerBlock = 256;
    int blocksPerGrid = (nSU * nch * nw + threadsPerBlock - 1) / threadsPerBlock;
    generate_data_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_data, d_noises, d_distances, nPU, nSU, nch, nw, beta, alpha, DistAmp, noi_pw);

    // 同步并获取结果
    double *data = new double[nSU * nch * nw];
    hipMemcpy(data, d_data, nSU * nch * nw * sizeof(double), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    // 处理结果并返回
    std::vector<std::vector<std::vector<double>>> result(nSU, std::vector<std::vector<double>>(nch, std::vector<double>(nw)));
    for (int suIdx = 0; suIdx < nSU; ++suIdx)
    {
        for (int chIdx = 0; chIdx < nch; ++chIdx)
        {
            for (int sampleIdx = 0; sampleIdx < nw; ++sampleIdx)
            {
                result[suIdx][chIdx][sampleIdx] = data[suIdx * nch * nw + chIdx * nw + sampleIdx];
            }
        }
    }

    // 清理设备内存
    hipFree(d_distances);
    hipFree(d_data);
    hipFree(d_noises);
    delete[] distances;
    delete[] noises;
    delete[] data;

    return result;
}
